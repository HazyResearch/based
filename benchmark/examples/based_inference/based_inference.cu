#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <mma.h>
using namespace nvcuda;

#include "src/global_warp_tile/warp_tile_abstract.cuh"
#include "src/reg_tile/register_tile.cuh" 
#include "src/reg_tile/register_frag.cuh"
#include "src/pyutils/torch_helpers.cuh"

// **** ASYNC INCLUDE *****
#include <cuda/pipeline>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

#include <ATen/cuda/HIPContext.h>

const int d_model =  64; 
const int d_state = 320;

template<typename H> __device__ float4* _f4p(H *x)  { return (float4* ) x;}
template<typename H> __device__ const float4* _f4pc(H *x)  { return (const float4* ) x;}
template<typename H> __device__ int _bank(H*x) { return ((uint64_t)x)/4 % 32; }


template <typename H, typename T>
__global__
void based_simple_ker(const T* __q, const T* __k, const T* __v, // single row of q, k, v of size q model.
                    T* __kv_state, T* __k_state,
                    T* __out) { 

    auto block_start = blockIdx.x;
    auto warpid = gwt::warp_id();
    auto lane   = gwt::laneid();
    const int workers  = 8; 
    const int nThreads = workers*gwt::WARP_SIZE;

    // Data size information
    const int kv_state_size = d_model * d_state;
    const int k_state_size  = d_state;

    const H *q_g = device_cast(__q)+block_start*d_state;
    const H *k_g = device_cast(__k)+block_start*d_state;
    const H *v_g = device_cast(__v)+block_start*d_model;
          H *kv_state_g = device_cast(__kv_state)+block_start*kv_state_size;
          H *k_state_g  = device_cast(__k_state)+block_start*k_state_size;
          H *num_g      = device_cast(__out)+block_start*d_model;

    // Setup the extended shared memory. We want to be 128 byte aligned.
    const int row_bytes   = d_model * sizeof(H);
    const int align_pad   = (row_bytes & 127) == 0 ? 0 : 128 - row_bytes & 127;
    const int buffer_rows = workers * 8; 
    const int buffer_size = (d_model + align_pad)*buffer_rows;
    assert(align_pad == 0); 

    // Use the simple shared memory 
    __shared__ alignas(alignof(float4)) H kv_state[2][buffer_size];
    __shared__ alignas(alignof(float4)) H q[d_state];
    __shared__ alignas(alignof(float4)) H k[d_state];
    __shared__ alignas(alignof(float4)) H k_state[d_state];
    __shared__ alignas(alignof(float4)) H v[d_model];
    __shared__ alignas(alignof(float4)) H num[d_model];
    __shared__ alignas(alignof(float4)) H num_help[workers][d_model];

    auto block = cooperative_groups::this_thread_block();
    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
    if (threadIdx.x == 0) {init(&barrier, block.size());}
     __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier_cheat;
    if (threadIdx.x == 0) {init(&barrier_cheat, block.size());}
    block.sync(); // need to make sure none calls before setup.
    
    // How many float4s -- these are 16 bytes.
    assert(d_state*sizeof(H) % sizeof(float4) == 0);
    assert(d_model*sizeof(H) % sizeof(float4) == 0);

    const auto d_state_shape = cuda::aligned_size_t<alignof(float4)>(d_state*sizeof(H)); 
    const auto d_model_shape = cuda::aligned_size_t<alignof(float4)>(d_model*sizeof(H)); 

    cuda::memcpy_async(block, q, q_g, d_state_shape, barrier_cheat);
    cuda::memcpy_async(block, k, k_g, d_state_shape, barrier_cheat);
    cuda::memcpy_async(block, k_state, k_state_g, d_state_shape, barrier_cheat);
    cuda::memcpy_async(block, v, v_g, d_model_shape, barrier_cheat);

    int tic = 0;
    int toc = 1;
    
    // Read the initial buffer slice of kv_state
    const auto buffer_row_shape = cuda::aligned_size_t<alignof(float4)>(buffer_rows*d_model*sizeof(H)); 
    cuda::memcpy_async(block, kv_state[tic], kv_state_g, buffer_row_shape, barrier);

    // Sum k to kstate and do kv. k_state += k
    barrier_cheat.arrive_and_wait(); // make sure q,k,v have arrived.
    for(auto i = threadIdx.x; i < d_state; i+=nThreads) {k_state[i] += k[i];}
    // store v across threads for the next phase.
    // assumes d_model fits in register and is a multiple of 32
    register H v_vals[d_model / gwt::WARP_SIZE];
    register H num_vals[d_model / gwt::WARP_SIZE];
    DEBUG_ONLY(assert(d_model % gwt::WARP_SIZE == 0);)

    auto j0 = 0;
    for(auto j = lane; j < d_model; j+=gwt::WARP_SIZE, ++j0) {
        v_vals[j0]   = v[j];
        num_vals[j0] = gwt::__typeconvert<float,H>(0.f);
    }
    // conduct the write back!
    __syncthreads(); // make sure all of k_state is written.
    for(auto i = threadIdx.x; i < d_state*sizeof(H)/sizeof(float4); i+=nThreads) {_f4p(k_state_g)[i] = _f4p(k_state)[i];}
    
    auto outer_batches = d_state / buffer_rows;
    auto extra_batch   = d_state % buffer_rows;
    
    const int bytes_per_batch = buffer_rows*d_model*sizeof(H);
    const int extra_bytes     = extra_batch*d_model*sizeof(H);
    const auto batch_shape    = cuda::aligned_size_t<alignof(float4)>(bytes_per_batch); 
    
    // Each row is of length d_model
    auto total_batches = (extra_batch > 0) ? outer_batches + 1 : outer_batches; 
    auto _buffer_rows  = buffer_rows; // will change if extra batches!
    // We iterate through each batch.
    // * We load the next batch asynchronously (into toc)
    // * We work on the current batch.
    // There is some cleanup about the extra batch.
    for(auto ob = 0; ob < total_batches; ob++, tic ^=1, toc ^=1) {
        auto cur_batch_idx  = ob       * buffer_rows * d_model;
        auto next_batch_idx = (ob + 1) * buffer_rows * d_model;

        barrier.arrive_and_wait(); // wait on the work buffer to be free
        if(ob + 1 < outer_batches) {// if there is more work fetch the next one.
            cuda::memcpy_async(block, kv_state[toc], 
                        kv_state_g + next_batch_idx, 
                        batch_shape , barrier);
        } else {// last batch!
            if(extra_batch > 0) {
                // NOTE: dmodel must be aligned for this to be true 
                DEBUG_ONLY(assert(d_model * sizeof(H) * extra_batch % sizeof(float4) == 0);)
                const auto extra_batch_shape = cuda::aligned_size_t<alignof(float4)>(extra_bytes); 
                cuda::memcpy_async(block, kv_state[toc], 
                        kv_state_g + next_batch_idx, 
                        extra_batch_shape, barrier); 
            }
        }
        // end load.

        _buffer_rows = (ob == outer_batches) ? extra_batch : buffer_rows;
        
        // Here we compute our chunk of: kv_state += torch.einsum("f,d->df", k, v)
        // we also compute: num = torch.einsum("f,df->d", q, kv_state)
        //
        // Each warp grabs a single value of k, k[i] and q[i]
        //
        // * The warp loops across the the kv_state on row i.
        // * Above, we stored all of v in register (v_vals) namely
        //   v[j] is stored in thread "j % 32" in a register.
        // * After reading and writing p_kvs that row is updatd.
        // * we compute num in a partitioned way, it represents:
        //   q[i]*kv_store[i,j]

        __syncwarp();
        for(auto i = warpid; i < _buffer_rows; i += workers) {
            H k_val = k[ob*buffer_rows + i]; // broadcast this value to each thread in the warp.
            H q_val = q[ob*buffer_rows + i];
            auto p_kvs = kv_state[tic] + i*d_model; // pointer to the row
            auto j0 = 0;
            for(auto j = lane; j < d_model; j += gwt::WARP_SIZE, j0++) { 
                p_kvs[j]     += k_val*v_vals[j0];
                num_vals[j0] += q_val*p_kvs[j];    
            }
        }
        __syncthreads(); // make sure the work is complete, then write back the buffered rows
        auto _stores = (ob == outer_batches) ? (extra_bytes/sizeof(float4)) : (bytes_per_batch/sizeof(float4));
        for(auto j = threadIdx.x; j < _stores; j+=nThreads) {
            _f4p(kv_state_g + cur_batch_idx)[j] = _f4p(kv_state[tic])[j];
        }
    }
    
    // At the end of the loop, the threads hold fragments (shared on j and i) 
    // for q*KV i num_vals[j0]. 
    // We need to aggregate across the warps.
    __syncwarp();
    j0 = 0;
    for(auto j = lane; j < d_model; j+= gwt::WARP_SIZE, ++j0) {
        num_help[warpid][j] = num_vals[j0];
    }
    __syncthreads(); // num_help is done
    for(auto j = threadIdx.x; j < d_model; j+=nThreads) {
        H nj = num_help[0][j];
        #pragma unroll
        for(auto w = 1; w < workers; w++) { nj += num_help[w][j]; }
        num[j] = nj;
    }

    __syncthreads();
    for(auto j = threadIdx.x; j < (d_model * sizeof(H))/sizeof(float4); j+=nThreads) {_f4p(num_g)[j] = _f4p(num)[j];}
}

void 
based_step(torch::Tensor q, torch::Tensor k, torch::Tensor v, 
            torch::Tensor kv_state, torch::Tensor k_state, torch::Tensor out) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(kv_state);
    CHECK_INPUT(k_state);
    CHECK_INPUT(out);

    auto batch = q.size(0);
    TORCH_CHECK(batch == k.size(0) && batch == v.size(0) && batch == kv_state.size(0) && k_state.size(0) == batch && out.size(0) == batch, "Differing batch sizes?");
    TORCH_CHECK(q.size(1) == d_state, "Q is d_state?");
    TORCH_CHECK(k.size(1) == d_state, "K is d_state?");
    TORCH_CHECK(v.size(1) == d_model, "V is d_model?");
    TORCH_CHECK(kv_state.size(1) == d_state && kv_state.size(2) == d_model);
    TORCH_CHECK(k_state.size(1) == d_state, "k_state is d_state");
    TORCH_CHECK(out.size(1) == d_model, "out is d_model (the size of v)");

    const int workers = 8;
    auto _run = [&]<typename H, typename T>() {
        auto threads = workers * gwt::WARP_SIZE;
        // unsigned long mem_size  = workers * 8 * 2 * d_model * sizeof(H);   
        // printf("[based_inference] Requesting %lu bytes of memory for %d (%d) workers for %d batches\n", mem_size, workers, threads, batch);

        auto stream_wrapper = at::cuda::getCurrentCUDAStream(q.device().index());
        hipStream_t stream = stream_wrapper.stream();
        based_simple_ker<H,T><<<batch,threads,0,stream>>>(
                q.data_ptr<T>(), k.data_ptr<T>(), v.data_ptr<T>(),
                kv_state.data_ptr<T>(), k_state.data_ptr<T>(),
                out.data_ptr<T>());
    };
    DISPATCH(q, (_run.operator()<H,T>()););
}
